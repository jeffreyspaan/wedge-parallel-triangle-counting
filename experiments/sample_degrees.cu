/* Sample degrees for a base and reordered graph
 * Jeffrey Spaan, Kuan-Hsun Chen, David Bader, Ana-Lucia Varbanescu
 *
 * Built on the work and code of David Bader. See https://github.com/Bader-Research/triangle-counting/ and https://doi.org/10.1109/HPEC58863.2023.10363539
 *
 * See usage() for instructions.
 * 
 * Assumptions:
 *	- Target GPU is device 0.
 *	- Number of vertices < (uint32_max / 2).
 *	- Number of edges < (uint64_max / 2).
 *	- Number of wedges < uint64_max.
 * 	- Max degree < uint32_max
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <stdbool.h>
#include <sys/time.h>

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_sort.cuh>

#define CHECK_BOUNDS 1
#define RESET_DEVICE 0
#define BINSEARCH_CONSTANT 1

#define BINSEARCH_CONSTANT_LEVELS 12
#define BINSEARCH_CONSTANT_CACHE_SIZE ((1 << BINSEARCH_CONSTANT_LEVELS) - 1) // 2^levels - 1

#define UINT_t uint32_t
#define INT_t int32_t
#define ULONG_t uint64_t

#define max2(a,b) ((a)>(b)?(a):(b))
#define min2(a,b) ((a)<(b)?(a):(b))

#define checkCudaErrors(call)																 						\
	do {																																	\
		hipError_t err = call;																	 						\
		if (err != hipSuccess) {																 						\
			fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,	\
						 hipGetErrorString(err));																	\
			exit(EXIT_FAILURE);																		 						\
		}																												 						\
	} while (0)

enum preprocess_t { PREPROCESS_CPU = 0, PREPROCESS_GPU, PREPROCESS_GPU_CONSTRAINED};

typedef struct {
	UINT_t numVertices;
	ULONG_t numEdges;
	ULONG_t* rowPtr;
	UINT_t* colInd;
} BIG_GRAPH_TYPE;

typedef struct {
	UINT_t src;
	UINT_t dst;
} edge_t;

typedef struct {
	UINT_t id;
	UINT_t *edges;
	UINT_t num_edges;
} preprocess_vertex_t;

/*********
 *  CPU  *
 *********/

static void assert_malloc(const void *ptr) {
	if (ptr==NULL) {
		fprintf(stderr,"ERROR: failed to allocate host memory.\n");
		exit(EXIT_FAILURE);
	}
}

void usage() {
	printf("Sample degrees for a base and reordered graph\n\n");
	printf("Usage:\n\n");
	printf("Either one of these must be selected:\n");
	printf(" -m <filename>        [Input graph in Matrix Market format]\n");
	printf(" -e <filename>        [Input graph in edge list format]\n");
	printf("Optional arguments:\n");
	printf(" -s <num>             [Number of samples (default 5000)]\n");
	printf(" -z                   [Input graph is zero-indexed]\n");
	printf(" -p                   [Preprocessing style, 0:CPU, 1:GPU, 2:GPU low-memory (default)]\n");
	printf("\n");
	printf("Example:\n");
	printf("./sample_degrees -m Amazon0302.mtx -s 10\n");
	exit(EXIT_FAILURE);
}

static int compareInt_t(const void *a, const void *b) {
	UINT_t arg1 = *(const UINT_t *)a;
	UINT_t arg2 = *(const UINT_t *)b;
	if (arg1 < arg2) return -1;
	if (arg1 > arg2) return 1;
	return 0;
}

static int compareEdge_t(const void *a, const void *b) {
	edge_t arg1 = *(const edge_t *) a;
	edge_t arg2 = *(const edge_t *) b;
	if (arg1.src < arg2.src) return -1;
	if (arg1.src > arg2.src) return 1;
	if ((arg1.src == arg2.src) && (arg1.dst < arg2.dst)) return -1;
	if ((arg1.src == arg2.src) && (arg1.dst > arg2.dst)) return 1;
	return 0;
}

static int compare_vertex_degree_ascending(const void *a, const void *b) {
	preprocess_vertex_t arg1 = *(const preprocess_vertex_t *) a;
	preprocess_vertex_t arg2 = *(const preprocess_vertex_t *) b;
	if (arg1.num_edges < arg2.num_edges) return -1;
	if (arg1.num_edges > arg2.num_edges) return 1;
	return 0;
}

struct edge_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&, unsigned int&> operator()(edge_t& key) const {
    return {key.src, key.dst};
  }
};

struct preprocess_vertex_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&> operator()(preprocess_vertex_t& key) const {
    return {key.num_edges};
  }
};

edge_t *sort_edges_GPU(edge_t *d_in, edge_t *d_out, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<edge_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

preprocess_vertex_t *sort_vertices_GPU(preprocess_vertex_t *d_in, preprocess_vertex_t *d_out, const UINT_t num_vertices, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<preprocess_vertex_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

UINT_t *sort_colInd_GPU(ULONG_t *d_rowPtr, UINT_t *d_colInd_in, UINT_t *d_colInd_out, const UINT_t num_vertices, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<UINT_t> d_keys(d_colInd_in, d_colInd_out);
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_colInd_out;
	}
}


BIG_GRAPH_TYPE *read_graph(char *filename, bool matrix_market, bool zero_indexed, preprocess_t preprocess_style) {
	FILE *infile = fopen(filename, "r");
	if (infile == NULL) {
		fprintf(stderr, "ERROR: unable to open graph file.\n");
		usage();
	}

	BIG_GRAPH_TYPE *graph = (BIG_GRAPH_TYPE *) malloc(sizeof(BIG_GRAPH_TYPE));
	char line[256];

	/* Skip any header lines */
	do {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	} while (line[0] < '0' || line[0] > '9');

	/* Skip line if the file is in Matrix Market format. We do not use the given vertex/edge counts. */
	if (matrix_market) {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	}

	UINT_t vertex_count = 0;
	ULONG_t edge_count = 0;
	size_t size = 10240;
	edge_t* edges = (edge_t*) malloc(size * sizeof(edge_t));
	assert_malloc(edges);

	UINT_t max_vertex = 0;
	UINT_t v, w;

	if (sscanf(line, "%d %d\n", &v, &w) == 2) {
		do {
			if (edge_count >= size) {
				size += 10240;	
				edge_t *new_edges = (edge_t*) realloc(edges, size * sizeof(edge_t));
				assert_malloc(new_edges);
				edges = new_edges;
			}

			if ((!zero_indexed) && (v == 0 || w == 0)) {
				fprintf(stderr, "ERROR: zero vertex id detected but -z was not set.\n");
				usage();
			}

			v -= (zero_indexed ? 0 : 1);
			w -= (zero_indexed ? 0 : 1);

			/* Remove self-loops. */
			if (v != w) {
				max_vertex = max2(max_vertex, max2(v, w));

				/* v->w */
				edges[edge_count].src = v;
				edges[edge_count].dst = w;
				edge_count++;
				/* w->v */
				edges[edge_count].src = w;
				edges[edge_count].dst = v;
				edge_count++;
			}
		} while (fscanf(infile, "%d %d\n", &v, &w) == 2);
	}

	fclose(infile);

	vertex_count = max_vertex + 1;

	/* Sort edges (in order to remove duplicates). */
	if (preprocess_style != PREPROCESS_CPU) {
		edge_t *d_edges;
		edge_t *d_edges_alt;
		edge_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_edges, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMalloc((void **)&d_edges_alt, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMemcpy(d_edges, edges, edge_count * sizeof(edge_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, true);
		else
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, false);

		checkCudaErrors(hipMemcpy(edges, d_out, edge_count * sizeof(edge_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_edges));
		checkCudaErrors(hipFree(d_edges_alt));
	} else {
		qsort(edges, edge_count, sizeof(edge_t), compareEdge_t);
	}	
	
	ULONG_t *rowPtr = (ULONG_t *) calloc(vertex_count+1, sizeof(ULONG_t));
	assert_malloc(rowPtr);

	ULONG_t edge_count_no_dup = 1;

	edge_t lastedge;
	lastedge.src = edges[0].src;
	lastedge.dst = edges[0].dst;

	UINT_t *colInd = (UINT_t *) edges; /* colInd overwrites the edges array. Possible because sizeof(edge_t) > sizeof(UINT_t). */
	colInd[0] = lastedge.dst;
	rowPtr[lastedge.src + 1]++;

	/* Remove duplicate edges. */
	for (ULONG_t i=1; i<edge_count; i++) {
		if (compareEdge_t(&lastedge, &edges[i]) != 0) {
			colInd[edge_count_no_dup++] = edges[i].dst;
			rowPtr[edges[i].src + 1]++;
			lastedge.src = edges[i].src;
			lastedge.dst = edges[i].dst;
		}
	}

	/* Free excess memory from the colInd/edges array. */
	UINT_t *new_colInd = (UINT_t *) realloc(colInd, edge_count_no_dup * sizeof(UINT_t));

	for (UINT_t v=1; v<=vertex_count; v++) {
		rowPtr[v] += rowPtr[v-1];
	}

	graph->numVertices = vertex_count;
	graph->numEdges = edge_count_no_dup;
	graph->rowPtr = rowPtr;
	graph->colInd = new_colInd;

	return graph;
}

BIG_GRAPH_TYPE *preprocess_base(const BIG_GRAPH_TYPE *original_graph, preprocess_t preprocess_style) {
	BIG_GRAPH_TYPE *graph = (BIG_GRAPH_TYPE *) malloc(sizeof(BIG_GRAPH_TYPE));
	assert_malloc(graph);

	graph->numVertices = original_graph->numVertices;
	graph->numEdges = original_graph->numEdges/2;

	graph->rowPtr = (ULONG_t *) malloc((graph->numVertices + 1) * sizeof(ULONG_t));
	assert_malloc(graph->rowPtr);
	graph->colInd = (UINT_t *) malloc(graph->numEdges * sizeof(UINT_t));
	assert_malloc(graph->colInd);

	ULONG_t edge = 0;
	graph->rowPtr[0] = 0;
	UINT_t max_degree = 0;

	for (UINT_t v=0; v<graph->numVertices; v++) {
		graph->rowPtr[v+1] = graph->rowPtr[v];

		for (ULONG_t i=original_graph->rowPtr[v]; i<original_graph->rowPtr[v+1]; i++) {
			UINT_t w = original_graph->colInd[i];
			if (v < w) {
				graph->colInd[edge++] = w;
				graph->rowPtr[v+1]++;
			}
		}

		max_degree = max2(max_degree, (UINT_t) graph->rowPtr[v+1] - graph->rowPtr[v]);
	}

	// printf("max_degree=%u\n", max_degree);

	return graph;
}

BIG_GRAPH_TYPE *preprocess(const BIG_GRAPH_TYPE *original_graph, preprocess_t preprocess_style) {
	preprocess_vertex_t *vertices = (preprocess_vertex_t *) malloc(original_graph->numVertices * sizeof(preprocess_vertex_t));
	assert_malloc(vertices);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		vertices[v].id = v;
		vertices[v].edges = &original_graph->colInd[original_graph->rowPtr[v]];
		vertices[v].num_edges = original_graph->rowPtr[v+1] - original_graph->rowPtr[v];
	}

	if (preprocess_style != PREPROCESS_CPU) {
		preprocess_vertex_t *d_vertices;
		preprocess_vertex_t *d_vertices_alt;
		preprocess_vertex_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_vertices, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMalloc((void **)&d_vertices_alt, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMemcpy(d_vertices, vertices, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, true);
		else
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, false);

		checkCudaErrors(hipMemcpy(vertices, d_out, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_vertices));
		checkCudaErrors(hipFree(d_vertices_alt));
	} else {
		qsort(vertices, original_graph->numVertices, sizeof(preprocess_vertex_t), compare_vertex_degree_ascending);
	}

	UINT_t *reverse = (UINT_t *) malloc(original_graph->numVertices * sizeof(UINT_t));
	assert_malloc(reverse);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		reverse[vertices[v].id] = v;
	}

	BIG_GRAPH_TYPE *graph = (BIG_GRAPH_TYPE *) malloc(sizeof(BIG_GRAPH_TYPE));
	assert_malloc(graph);

	graph->numVertices = original_graph->numVertices;
	graph->numEdges = original_graph->numEdges/2;

	graph->rowPtr = (ULONG_t*) malloc((graph->numVertices + 1) * sizeof(ULONG_t));
	assert_malloc(graph->rowPtr);
	graph->colInd = (UINT_t*) malloc(graph->numEdges * sizeof(UINT_t));
	assert_malloc(graph->colInd);

	ULONG_t edge_count = 0;

	graph->rowPtr[0] = 0;

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		UINT_t new_degree = 0;

		for (UINT_t j=0; j<vertices[v].num_edges; j++) {
			UINT_t w = vertices[v].edges[j];
			UINT_t w_new = reverse[w];

			if (w_new > v) {
				graph->colInd[edge_count++] = w_new;
				new_degree++;
			}
		}

		graph->rowPtr[v+1] = graph->rowPtr[v] + new_degree;

		if (preprocess_style == PREPROCESS_CPU) {
			qsort(&graph->colInd[graph->rowPtr[v]], new_degree, sizeof(UINT_t), compareInt_t);
		}
	}

	free(vertices);
	free(reverse);

	if (preprocess_style != PREPROCESS_CPU) {
		ULONG_t *d_rowPtr;
		UINT_t *d_colInd;
		UINT_t *d_colInd_alt;
		UINT_t *d_colInd_out;

		checkCudaErrors(hipMalloc((void **)&d_rowPtr, (graph->numVertices+1) * sizeof(ULONG_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd_alt, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMemcpy(d_rowPtr, graph->rowPtr, (graph->numVertices+1) * sizeof(ULONG_t), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_colInd, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, true);
		else
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, false);

		checkCudaErrors(hipMemcpy(graph->colInd, d_colInd_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_rowPtr));
		checkCudaErrors(hipFree(d_colInd));
		checkCudaErrors(hipFree(d_colInd_alt));
	}

	return graph;
}

void free_graph(BIG_GRAPH_TYPE *graph) {
	free(graph->rowPtr);
	free(graph->colInd);
	free(graph);
}

int main(int argc, char **argv) {
	char *graph_filename = NULL;
	bool graph_mm = false;
	bool graph_zero_indexed = false;
	UINT_t samples = 5000;

	/* Default: use lightweight GPU-based preprocessing (worst case ~ m*8 device memory). */
	preprocess_t preprocess_style = PREPROCESS_GPU_CONSTRAINED;

	while ((argc > 1) && (argv[1][0] == '-')) {
		switch (argv[1][1]) {
			case 'm':
				graph_mm = true;
			case 'e':
				if (argc < 3) usage();
				graph_filename = argv[2];
				if (graph_filename == NULL) usage();
				argv+=2;
				argc-=2;
				break;
			case 'z':
				graph_zero_indexed = true;
				argv++;
				argc--;
				break;
			case 's':
				if (argc < 3) usage();
				samples = atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
			case 'p':
				if (argc < 3) usage();
				if (atoi(argv[2]) < PREPROCESS_CPU || atoi(argv[2]) > PREPROCESS_GPU_CONSTRAINED) usage();
				preprocess_style = (preprocess_t) atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
		}
	}

	if (graph_filename == NULL) usage();

	BIG_GRAPH_TYPE *original_graph = read_graph(graph_filename, graph_mm, graph_zero_indexed, preprocess_style);
	BIG_GRAPH_TYPE *base_graph = preprocess_base(original_graph, preprocess_style);

	UINT_t base_max_degree = 0;
	ULONG_t base_num_wedges = 0;

	for (UINT_t v=0; v<base_graph->numVertices; v++) {
		UINT_t degree = (UINT_t) (base_graph->rowPtr[v+1] - base_graph->rowPtr[v]);
		base_max_degree = max2(base_max_degree, degree);
		base_num_wedges += (degree * (degree - 1)) / 2;
	}

	printf("%-16s %-16s\n", "preprocessing", "base");
	printf("%-16s %-16u\n", "num_vertices", base_graph->numVertices);
	printf("%-16s %-32lu\n", "num_wedges", base_num_wedges);
	printf("%-16s %-16u\n", "max_degree", base_max_degree);

	UINT_t base_stop = (base_graph->numVertices < samples) ? base_graph->numVertices : ((UINT_t) (base_graph->numVertices / samples)) * samples;
	UINT_t base_step = (base_graph->numVertices < samples) ? 1 : ((UINT_t) (base_graph->numVertices / samples));

	for (UINT_t v=0, i=0; v<base_stop; v+=base_step, i++) {
		printf("%-16u %-16u\n", i, (UINT_t) (base_graph->rowPtr[v+1] - base_graph->rowPtr[v]));
	}

	free_graph(base_graph);
	
	BIG_GRAPH_TYPE *reordered_graph = preprocess(original_graph, preprocess_style);

	UINT_t reordered_max_degree = 0;
	ULONG_t reordered_num_wedges = 0;

	for (UINT_t v=0; v<reordered_graph->numVertices; v++) {
		UINT_t degree = (UINT_t) (reordered_graph->rowPtr[v+1] - reordered_graph->rowPtr[v]);
		reordered_max_degree = max2(reordered_max_degree, degree);
		reordered_num_wedges += (degree * (degree - 1)) / 2;
	}

	printf("%-16s %-16s\n", "preprocessing", "reordered");
	printf("%-16s %-16u\n", "num_vertices", reordered_graph->numVertices);
	printf("%-16s %-32lu\n", "num_wedges", reordered_num_wedges);
	printf("%-16s %-16u\n", "max_degree", reordered_max_degree);

	UINT_t reordered_stop = (reordered_graph->numVertices < samples) ? reordered_graph->numVertices : ((UINT_t) (reordered_graph->numVertices / samples)) * samples;
	UINT_t reordered_step = (reordered_graph->numVertices < samples) ? 1 : ((UINT_t) (reordered_graph->numVertices / samples));

	for (UINT_t v=0, i=0; v<reordered_stop; v+=reordered_step, i++) {
		printf("%-16u %-16u\n", i, (UINT_t) (reordered_graph->rowPtr[v+1] - reordered_graph->rowPtr[v]));
	}

	free_graph(reordered_graph);
	free_graph(original_graph);

	return EXIT_SUCCESS;
}
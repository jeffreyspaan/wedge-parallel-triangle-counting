#include "hip/hip_runtime.h"
/* Edge-parallel Triangle Counting
 * Jeffrey Spaan, Ana-Lucia Varbanescu, Kuan Chen.
 *
 * Built on the work and code of David Bader. See https://github.com/Bader-Research/triangle-counting/ and https://doi.org/10.1109/HPEC58863.2023.10363539
 *
 * See usage() for instructions.
 * 
 * Assumptions:
 *	- Target GPU is device 0.
 *	- Number of vertices < (uint32_t max / 2).
 *	- Number of edges < uint32_t max.
 *	- Number of wedges < uint64_t max.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <stdbool.h>
#include <sys/time.h>

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_sort.cuh>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define RESET_DEVICE 0
#define BINSEARCH_CONSTANT 1

#define BINSEARCH_CONSTANT_LEVELS 12
#define BINSEARCH_CONSTANT_CACHE_SIZE ((1 << BINSEARCH_CONSTANT_LEVELS) - 1) // 2^levels - 1

#define UINT_t uint32_t
#define INT_t int32_t
#define ULONG_t uint64_t

#define max2(a,b) ((a)>(b)?(a):(b))
#define min2(a,b) ((a)<(b)?(a):(b))

static struct timeval	tp;
static struct timezone tzp;

#define get_seconds()	 (gettimeofday(&tp, &tzp), \
												(double)tp.tv_sec + (double)tp.tv_usec / 1000000.0)

#define checkCudaErrors(call)																 						\
	do {																																	\
		hipError_t err = call;																	 						\
		if (err != hipSuccess) {																 						\
			fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,	\
						 hipGetErrorString(err));																	\
			exit(EXIT_FAILURE);																		 						\
		}																												 						\
	} while (0)

enum preprocess_t { PREPROCESS_CPU = 0, PREPROCESS_GPU, PREPROCESS_GPU_CONSTRAINED};

enum wedge_style_t { WEDGE_STYLE_OUTGOING = 0, WEDGE_STYLE_ARROW, WEDGE_STYLE_MIXED};

typedef struct {
	UINT_t numVertices;
	UINT_t numEdges;
	UINT_t* rowPtr;
	UINT_t* colInd;
} GRAPH_TYPE;

typedef struct {
	UINT_t src;
	UINT_t dst;
} edge_t;

typedef struct {
	UINT_t id;
	UINT_t new_id;
	UINT_t num_edges;
	UINT_t *edges;
} preprocess_vertex_t;

typedef struct {
	double copy;
	double exec;
} GPU_time;

/*********
 *	GPU	*
 *********/

#if BINSEARCH_CONSTANT
__constant__ UINT_t c_binary_search_cache[BINSEARCH_CONSTANT_CACHE_SIZE];
#endif

__device__ INT_t linearSearch_GPU(const UINT_t* list, const UINT_t start, const UINT_t end, const UINT_t target) {
	for (UINT_t i=start; i<end; i++) {
		if (list[i] == target) {
			return i;
		} else if (list[i] > target) {
			break;
		}
	}

	return -1;
}

__device__ INT_t binary_search_GPU(const UINT_t* list, const UINT_t start, const UINT_t end, const UINT_t target) {
	UINT_t s=start, e=end, mid;
	while (s < e) {
		mid = (s + e) >> 1;
		if (list[mid] == target)
			return mid;

		if (list[mid] < target)
			s = mid + 1;
		else
			e = mid;
	}
	return -1;
}


__device__ UINT_t binary_search_closest_GPU(const UINT_t* list, const UINT_t start, const UINT_t end, const UINT_t target) {
	/* Finds the index of the rightmost closest value smaller or equal than target, e.g.,
	 * for target 1 and list=[0,0,0,2,2,2] it returns 2,
	 * for target 2 and list=[0,0,0,2,2,2] it returns 5.
	 * Assumes list[0]=0
	 * Assumes end-1 <= UINT_MAX/2
	 */

	UINT_t s=start, e=end, mid;
	while (s < e) {
		mid = (s + e) >> 1;

		if (list[mid] < target+1) {
			s = mid + 1;
		} else {
			e = mid;
		}
	}
	
	return max2(start, (s > 0) ? s-1: 0);
}

#if BINSEARCH_CONSTANT
__device__ UINT_t binary_search_closest_constant_GPU(const UINT_t *list, const UINT_t start, const UINT_t end, const UINT_t target) {
	/* Finds the index of the rightmost closest value smaller or equal than target.
	 * Uses constant memory for the first BINSEARCH_CONSTANT_LEVELS levels.
	 */
	UINT_t mid;

	UINT_t g_s = start;
	UINT_t g_e = end;
	UINT_t g_mid;

	UINT_t c_index = 0;

	#pragma unroll
	for (UINT_t iter=0; iter<BINSEARCH_CONSTANT_LEVELS; iter++) {
		mid = c_binary_search_cache[c_index];
		g_mid = (g_s+g_e) >> 1;

		c_index *= 2;
		c_index += 1;

		if (mid < target+1) {
			c_index += 1;
			g_s = g_mid+1;
		} else {
			g_e = g_mid;
		}
	}

	g_s = max2(start, (g_s > 0) ? g_s-1 : 0);
	return binary_search_closest_GPU(list, g_s, g_e, target);
}
#endif

__global__ void tc_edge_outgoing_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const UINT_t num_edges, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const ULONG_t tid = ((ULONG_t) blockIdx.x * blockDim.x + threadIdx.x);
	ULONG_t count = 0;

	if (tid < num_edges) {
		UINT_t w = g_Ai[tid];
#if BINSEARCH_CONSTANT
		UINT_t v = binary_search_closest_constant_GPU(g_Ap, 0, num_vertices, tid);
#else
		UINT_t v = binary_search_closest_GPU(g_Ap, 0, num_vertices, tid);
#endif

		UINT_t ve = g_Ap[v+1];

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		for (UINT_t i=tid+1; i<ve; i++) {
			UINT_t u = g_Ai[i];

			if (w >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
				ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-w) * (ULONG_t) ((num_vertices-w)-1)) >> 1)) + u - w - 1;

	#if UINT_WIDTH == 32
				bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
	#else
				bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
	#endif
				if (found) {
					count++;
				}
			} else {
				if (binary_search_GPU(g_Ai, wb, we, u) >= 0) {
					count++;
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}

__global__ void tc_edge_arrow_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const UINT_t num_edges, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const ULONG_t tid = ((ULONG_t) blockIdx.x * blockDim.x + threadIdx.x);
	ULONG_t count = 0;

	if (tid < num_edges) {
		UINT_t w = g_Ai[tid];
#if BINSEARCH_CONSTANT
		UINT_t v = binary_search_closest_constant_GPU(g_Ap, 0, num_vertices, tid);
#else
		UINT_t v = binary_search_closest_GPU(g_Ap, 0, num_vertices, tid);
#endif

		UINT_t vb = g_Ap[v];
		UINT_t ve = g_Ap[v+1];

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		for (UINT_t i=wb; i<we; i++) {
			UINT_t u = g_Ai[i];

			if (v >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
				ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-v) * (ULONG_t) ((num_vertices-v)-1)) >> 1)) + u - v - 1;

	#if UINT_WIDTH == 32
				bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
	#else
				bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
	#endif
				if (found) {
					count++;
				}
			} else {
				/* Note: in general, searching from vb seems to be faster than from tid+1. */
				if (binary_search_GPU(g_Ai, vb, ve, u) >= 0) {
					count++;
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}


__global__ void tc_edge_mixed_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const UINT_t num_edges, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const ULONG_t tid = ((ULONG_t) blockIdx.x * blockDim.x + threadIdx.x);
	ULONG_t count = 0;

	if (tid < num_edges) {
		UINT_t w = g_Ai[tid];
#if BINSEARCH_CONSTANT
		UINT_t v = binary_search_closest_constant_GPU(g_Ap, 0, num_vertices, tid);
#else
		UINT_t v = binary_search_closest_GPU(g_Ap, 0, num_vertices, tid);
#endif

		UINT_t vb = g_Ap[v];
		UINT_t ve = g_Ap[v+1];

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		UINT_t size_v = ve-(tid+1);
		UINT_t size_w = we-wb;

		if (size_v <= size_w) {
			for (UINT_t i=tid+1; i<ve; i++) {
				UINT_t u = g_Ai[i];

				if (w >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
					ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-w) * (ULONG_t) ((num_vertices-w)-1)) >> 1)) + u - w - 1;

		#if UINT_WIDTH == 32
					bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
		#else
					bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
		#endif
					if (found) {
						count++;
					}
				} else {
					if (binary_search_GPU(g_Ai, wb, we, u) >= 0) {
						count++;
					}
				}
			}
		} else {
			for (UINT_t i=wb; i<we; i++) {
				UINT_t u = g_Ai[i];

				if (v >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
					ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-v) * (ULONG_t) ((num_vertices-v)-1)) >> 1)) + u - v - 1;

		#if UINT_WIDTH == 32
					bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
		#else
					bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
		#endif
					if (found) {
						count++;
					}
				} else {
					/* Note: in general, searching from vb seems to be faster than from tid+1. */
					if (binary_search_GPU(g_Ai, vb, ve, u) >= 0) {
						count++;
					}
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}

/*********
 *	CPU	*
 *********/

static void assert_malloc(const void *ptr) {
	if (ptr==NULL) {
		fprintf(stderr,"ERROR: failed to allocate host memory.\n");
		exit(EXIT_FAILURE);
	}
}

void build_binary_search_cache(UINT_t *src, UINT_t *cache, UINT_t level, UINT_t max_level, UINT_t i, UINT_t s, UINT_t e) {
	if (level < max_level) {
		UINT_t mid = (s + e) / 2;
		cache[i] = src[mid];
		build_binary_search_cache(src, cache, level+1, max_level, i*2 + 1, s, mid);
		build_binary_search_cache(src, cache, level+1, max_level, i*2 + 2, mid+1, e);
	}
}

ULONG_t tc_edge_GPU(const GRAPH_TYPE *graph, UINT_t adjacency_matrix_len, wedge_style_t style, GPU_time *t) {
	UINT_t *d_Ap;
	UINT_t *d_Ai;
	UINT_t *d_adjacency_matrix;
	ULONG_t *d_total_count;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));

	/* Store all existing edges of the vertices from n-adjacency_matrix_len to n in an upper triangluar adjacency matrix. */
	ULONG_t adjacency_matrix_size = (((ULONG_t) adjacency_matrix_len) * ((ULONG_t) adjacency_matrix_len - 1)) / 2;
	UINT_t *h_adjacency_matrix = (UINT_t *) calloc(adjacency_matrix_size/UINT_WIDTH, sizeof(UINT_t));
	assert_malloc(h_adjacency_matrix);

	for (UINT_t v=graph->numVertices - min2(graph->numVertices, adjacency_matrix_len); v<graph->numVertices; v++) {
		for (UINT_t i=graph->rowPtr[v]; i<graph->rowPtr[v+1]; i++) {
			UINT_t w = graph->colInd[i];

			ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (graph->numVertices-v) * (ULONG_t) ((graph->numVertices-v)-1)) / 2)) + w - v - 1;

#if UINT_WIDTH == 32
			h_adjacency_matrix[adjacency_i >> 5] |= (1 << (adjacency_i & 31));
#else
			h_adjacency_matrix[adjacency_i / UINT_WIDTH] |= (1 << (adjacency_i % UINT_WIDTH));
#endif
		}
	}

#if BINSEARCH_CONSTANT
	UINT_t *h_rowPtr_cache = (UINT_t *) malloc(BINSEARCH_CONSTANT_CACHE_SIZE * sizeof(UINT_t));
	assert_malloc(h_rowPtr_cache);
	build_binary_search_cache(graph->rowPtr, h_rowPtr_cache, 0, BINSEARCH_CONSTANT_LEVELS, 0, 0, graph->numVertices);
#endif

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_adjacency_matrix, (adjacency_matrix_size/32) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_total_count, 1 * sizeof(ULONG_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_adjacency_matrix, h_adjacency_matrix, (adjacency_matrix_size/32) * sizeof(UINT_t), hipMemcpyHostToDevice));
	
#if BINSEARCH_CONSTANT
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_binary_search_cache), h_rowPtr_cache, BINSEARCH_CONSTANT_CACHE_SIZE * sizeof(UINT_t)));
#endif

	checkCudaErrors(hipMemset(d_total_count, 0, 1 * sizeof(ULONG_t)));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 128;
	ULONG_t num_blocks = (graph->numEdges / num_threads) + 1;

	if (num_blocks > (((ULONG_t) 1 << 31)-1)*num_threads) {
		fprintf(stderr, "ERROR: maximum grid size reached.\n");
		exit(EXIT_FAILURE);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipEventRecord(GPU_exec_start));


	if (style == WEDGE_STYLE_OUTGOING)
		tc_edge_outgoing_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, graph->numEdges, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);
	else if (style == WEDGE_STYLE_ARROW)
		tc_edge_arrow_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, graph->numEdges, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);
	else if (style == WEDGE_STYLE_MIXED)
		tc_edge_mixed_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, graph->numEdges, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);


	checkCudaErrors(hipEventRecord(GPU_exec_stop));
	checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
	t->exec += GPU_exec_elapsed;

	ULONG_t h_total_count = 0;
	checkCudaErrors(hipMemcpy(&h_total_count, d_total_count, 1 * sizeof(ULONG_t), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_adjacency_matrix));
	checkCudaErrors(hipFree(d_total_count));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_adjacency_matrix);

#if BINSEARCH_CONSTANT
	free(h_rowPtr_cache);
#endif
	
#if RESET_DEVICE
	checkCudaErrors(hipDeviceReset());
#endif
	return h_total_count;
}

void usage() {
	printf("Edge-parallel Triangle Counting\n\n");
	printf("Usage:\n\n");
	printf("Either one of these must be selected:\n");
	printf(" -m <filename>	[Input graph in Matrix Market format]\n");
	printf(" -e <filename>	[Input graph in edge list format]\n");
	printf("Required arguments:\n");
	printf(" -a <num>				[Adjacency matrix length] (must be divisble by 32)\n");
	printf("Optional arguments:\n");
	printf(" -w <num>				[Wedge style, 0:outgoing (default), 1:arrow, 2:mixed]\n");
	printf(" -l <num>				[Loop count]\n");
	printf(" -z							[Input graph is zero-indexed]\n");
	printf(" -p							[Preprocessing style, 0:CPU, 1:GPU, 2:GPU low-memory (default)]\n");
	printf("\n");
	printf("Example:\n");
	printf("./tc_edge -m ../Amazon0302.mtx -a 8192 -l 10\n");
	exit(EXIT_FAILURE);
}


static int compareInt_t(const void *a, const void *b) {
	UINT_t arg1 = *(const UINT_t *)a;
	UINT_t arg2 = *(const UINT_t *)b;
	if (arg1 < arg2) return -1;
	if (arg1 > arg2) return 1;
	return 0;
}

static int compareEdge_t(const void *a, const void *b) {
	edge_t arg1 = *(const edge_t *) a;
	edge_t arg2 = *(const edge_t *) b;
	if (arg1.src < arg2.src) return -1;
	if (arg1.src > arg2.src) return 1;
	if ((arg1.src == arg2.src) && (arg1.dst < arg2.dst)) return -1;
	if ((arg1.src == arg2.src) && (arg1.dst > arg2.dst)) return 1;
	return 0;
}

static int compare_vertex_degree_ascending(const void *a, const void *b) {
	preprocess_vertex_t arg1 = *(const preprocess_vertex_t *) a;
	preprocess_vertex_t arg2 = *(const preprocess_vertex_t *) b;
	if (arg1.num_edges < arg2.num_edges) return -1;
	if (arg1.num_edges > arg2.num_edges) return 1;
	return 0;
}

struct edge_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&, unsigned int&> operator()(edge_t& key) const {
    return {key.src, key.dst};
  }
};

struct preprocess_vertex_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&> operator()(preprocess_vertex_t& key) const {
    return {key.num_edges};
  }
};

edge_t *sort_edges_GPU(edge_t *d_in, edge_t *d_out, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<edge_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

preprocess_vertex_t *sort_vertices_GPU(preprocess_vertex_t *d_in, preprocess_vertex_t *d_out, const UINT_t num_vertices, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<preprocess_vertex_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

UINT_t *sort_colInd_GPU(UINT_t *d_rowPtr, UINT_t *d_colInd_in, UINT_t *d_colInd_out, const UINT_t num_vertices, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<UINT_t> d_keys(d_colInd_in, d_colInd_out);
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_colInd_out;
	}
}


GRAPH_TYPE *read_graph(char *filename, bool matrix_market, bool zero_indexed, preprocess_t preprocess_style) {
	FILE *infile = fopen(filename, "r");
	if (infile == NULL) {
		printf("ERROR: unable to open graph file.\n");
		usage();
	}

	GRAPH_TYPE *graph = (GRAPH_TYPE *) malloc(sizeof(GRAPH_TYPE));
	char line[256];

	/* Skip any header lines */
	do {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	} while (line[0] < '0' || line[0] > '9');

	/* Skip line if the file is in Matrix Market format. We do not use the given vertex/edge counts. */
	if (matrix_market) {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	}

	UINT_t vertex_count = 0;
	UINT_t edge_count = 0;
	size_t size = 10240;
	edge_t* edges = (edge_t*) malloc(size * sizeof(edge_t));
	assert_malloc(edges);

	UINT_t max_vertex = 0;
	UINT_t v, w;

	if (sscanf(line, "%d %d\n", &v, &w) == 2) {
		do {
			if (edge_count >= size) {
				size += 10240;	
				edge_t *new_edges = (edge_t*) realloc(edges, size * sizeof(edge_t));
				assert_malloc(new_edges);
				edges = new_edges;
			}

			if ((!zero_indexed) && (v == 0 || w == 0)) {
				fprintf(stderr, "ERROR: zero vertex id detected but -z was not set.\n");
				usage();
			}

			v -= (zero_indexed ? 0 : 1);
			w -= (zero_indexed ? 0 : 1);

			/* Remove self-loops. */
			if (v != w) {
				max_vertex = max2(max_vertex, max2(v, w));

				/* v->w */
				edges[edge_count].src = v;
				edges[edge_count].dst = w;
				edge_count++;
				/* w->v */
				edges[edge_count].src = w;
				edges[edge_count].dst = v;
				edge_count++;
			}
		} while (fscanf(infile, "%d %d\n", &v, &w) == 2);
	}

	fclose(infile);

	vertex_count = max_vertex + 1;

	/* Sort edges (in order to remove duplicates). */
	if (preprocess_style != PREPROCESS_CPU) {
		edge_t *d_edges;
		edge_t *d_edges_alt;
		edge_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_edges, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMalloc((void **)&d_edges_alt, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMemcpy(d_edges, edges, edge_count * sizeof(edge_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, true);
		else
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, false);

		checkCudaErrors(hipMemcpy(edges, d_out, edge_count * sizeof(edge_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_edges));
		checkCudaErrors(hipFree(d_edges_alt));
	} else {
		qsort(edges, edge_count, sizeof(edge_t), compareEdge_t);
	}	
	
	UINT_t *rowPtr = (UINT_t *) calloc(vertex_count+1, sizeof(UINT_t));
	assert_malloc(rowPtr);

	UINT_t edge_count_no_dup = 1;

	edge_t lastedge;
	lastedge.src = edges[0].src;
	lastedge.dst = edges[0].dst;

	UINT_t *colInd = (UINT_t *) edges; /* colInd overwrites the edges array. Possible because sizeof(edge_t) > sizeof(UINT_t). */
	colInd[0] = lastedge.dst;
	rowPtr[lastedge.src + 1]++;

	/* Remove duplicate edges. */
	for (UINT_t i=1; i<edge_count; i++) {
		if (compareEdge_t(&lastedge, &edges[i]) != 0) {
			colInd[edge_count_no_dup++] = edges[i].dst;
			rowPtr[edges[i].src + 1]++;
			lastedge.src = edges[i].src;
			lastedge.dst = edges[i].dst;
		}
	}

	/* Free excess memory from the colInd/edges array. */
	UINT_t *new_colInd = (UINT_t *) realloc(colInd, edge_count_no_dup * sizeof(UINT_t));

	for (UINT_t v=1; v<=vertex_count; v++) {
		rowPtr[v] += rowPtr[v-1];
	}

	graph->numVertices = vertex_count;
	graph->numEdges = edge_count_no_dup;
	graph->rowPtr = rowPtr;
	graph->colInd = new_colInd;

	return graph;
}

GRAPH_TYPE *preprocess(const GRAPH_TYPE *original_graph, preprocess_t preprocess_style) {
	preprocess_vertex_t *vertices = (preprocess_vertex_t *) malloc(original_graph->numVertices * sizeof(preprocess_vertex_t));
	assert_malloc(vertices);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		vertices[v].id = v;
		vertices[v].edges = &original_graph->colInd[original_graph->rowPtr[v]];
		vertices[v].num_edges = original_graph->rowPtr[v+1] - original_graph->rowPtr[v];
	}

	if (preprocess_style != PREPROCESS_CPU) {
		preprocess_vertex_t *d_vertices;
		preprocess_vertex_t *d_vertices_alt;
		preprocess_vertex_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_vertices, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMalloc((void **)&d_vertices_alt, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMemcpy(d_vertices, vertices, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, true);
		else
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, false);

		checkCudaErrors(hipMemcpy(vertices, d_out, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_vertices));
		checkCudaErrors(hipFree(d_vertices_alt));
	} else {
		qsort(vertices, original_graph->numVertices, sizeof(preprocess_vertex_t), compare_vertex_degree_ascending);
	}

	UINT_t *reverse = (UINT_t *) malloc(original_graph->numVertices * sizeof(UINT_t));
	assert_malloc(reverse);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		reverse[vertices[v].id] = v;
	}

	GRAPH_TYPE *graph = (GRAPH_TYPE *) malloc(sizeof(GRAPH_TYPE));
	assert_malloc(graph);

	graph->numVertices = original_graph->numVertices;
	graph->numEdges = original_graph->numEdges/2;

	graph->rowPtr = (UINT_t*) malloc((graph->numVertices + 1) * sizeof(UINT_t));
	assert_malloc(graph->rowPtr);
	graph->colInd = (UINT_t*) malloc(graph->numEdges * sizeof(UINT_t));
	assert_malloc(graph->colInd);

	UINT_t edge_count = 0;

	graph->rowPtr[0] = 0;

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		UINT_t new_degree = 0;

		for (INT_t j=0; j<vertices[v].num_edges; j++) {
			UINT_t w = vertices[v].edges[j];
			UINT_t w_new = reverse[w];

			if (w_new > v) {
				graph->colInd[edge_count++] = w_new;
				new_degree++;
			}
		}

		graph->rowPtr[v+1] = graph->rowPtr[v] + new_degree;

		if (preprocess_style == PREPROCESS_CPU) {
			qsort(&graph->colInd[graph->rowPtr[v]], new_degree, sizeof(UINT_t), compareInt_t);
		}
	}

	free(vertices);
	free(reverse);

	if (preprocess_style != PREPROCESS_CPU) {
		UINT_t *d_rowPtr;
		UINT_t *d_colInd;
		UINT_t *d_colInd_alt;
		UINT_t *d_colInd_out;

		checkCudaErrors(hipMalloc((void **)&d_rowPtr, (graph->numVertices+1) * sizeof(UINT_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd_alt, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMemcpy(d_rowPtr, graph->rowPtr, (graph->numVertices+1) * sizeof(UINT_t), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_colInd, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, true);
		else
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, false);

		checkCudaErrors(hipMemcpy(graph->colInd, d_colInd_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_rowPtr));
		checkCudaErrors(hipFree(d_colInd));
		checkCudaErrors(hipFree(d_colInd_alt));
	}

	return graph;
}

void free_graph(GRAPH_TYPE *graph) {
	free(graph->rowPtr);
	free(graph->colInd);
	free(graph);
}

void print_degrees(GRAPH_TYPE *graph, const char *filename, UINT_t num, bool oneify) {
	FILE *outfile = fopen(filename, "w");

	printf("n=%u step=%u\n", graph->numVertices, max2(1,(graph->numVertices / num)));

	for (UINT_t v=0; v<graph->numVertices; v += max2(1,(graph->numVertices / num))) {
		if (oneify) {
			UINT_t degree = 0;
			for (UINT_t i=graph->rowPtr[v]; i<graph->rowPtr[v+1]; i++) {
				if (graph->colInd[i] > v)
					degree++;
			}
			fprintf(outfile, "%u %u\n", v, degree);
		} else {
			fprintf(outfile, "%u %u\n", v, graph->rowPtr[v+1]-graph->rowPtr[v]);
		}
	}

	fclose(outfile);
}

int main(int argc, char **argv) {
	char *graph_filename = NULL;
	bool graph_mm = false;
	bool graph_zero_indexed = false;
	UINT_t loop_cnt = 1;

	/* Default: use lightweight GPU-based preprocessing (worst case ~ m*8 device memory). */
	preprocess_t preprocess_style = PREPROCESS_GPU_CONSTRAINED;

	wedge_style_t style = WEDGE_STYLE_OUTGOING;
	UINT_t adjacency_matrix_len = 0;

	while ((argc > 1) && (argv[1][0] == '-')) {
		switch (argv[1][1]) {
			case 'm':
				graph_mm = true;
			case 'e':
				if (argc < 3) usage();
				graph_filename = argv[2];
				if (graph_filename == NULL) usage();
				argv+=2;
				argc-=2;
				break;
			case 'z':
				graph_zero_indexed = true;
				argv++;
				argc--;
				break;
			case 'w':
				if (argc < 3) usage();
				if (atoi(argv[2]) < WEDGE_STYLE_OUTGOING || atoi(argv[2]) > WEDGE_STYLE_MIXED) usage();
				style = (wedge_style_t) atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
			case 'a':
				/* Adjacency matrix lengths:
				 *	 8192 ~= 4MiB
				 *	 16384 ~= 16MiB
				 *	 32768 ~= 64MiB
				 *	 65536 ~= 265MiB
				 *	 131072 ~= 1024MiB
				 */
				if (argc < 3) usage();
				adjacency_matrix_len = atoi(argv[2]);
				if (adjacency_matrix_len % 32 != 0) usage();
				argv+=2;
				argc-=2;
				break;
			case 'l':
				if (argc < 3) usage();
				loop_cnt = atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
			case 'p':
				if (argc < 3) usage();
				if (atoi(argv[2]) < PREPROCESS_CPU || atoi(argv[2]) > PREPROCESS_GPU_CONSTRAINED) usage();
				preprocess_style = (preprocess_t) atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
		}
	}

	if (graph_filename == NULL) usage();

	GRAPH_TYPE *original_graph = read_graph(graph_filename, graph_mm, graph_zero_indexed, preprocess_style);
	double t_preprocessing = get_seconds();
	GRAPH_TYPE *graph = preprocess(original_graph, preprocess_style);
	t_preprocessing = get_seconds() - t_preprocessing;
	free_graph(original_graph);

	printf("%-60s %16s %16s %16s %16s %16s %16s %16s %16s %16s %16s\n",
		"graph", "n", "m", "a", "wedge style", "triangles", "prepro (s)", "GPU copy (s)", "GPU exec (s)", "GPU total (s)", "CPU+GPU (s)");

	bool warmed_up = false;

	for (UINT_t i=0; i<(loop_cnt+1); i++) {
		double t_cpu = get_seconds();
		GPU_time t_gpu = { .copy=0.0, .exec=0.0 };

		ULONG_t triangles = tc_edge_GPU(graph, adjacency_matrix_len, style, &t_gpu);

		t_cpu = get_seconds() - t_cpu;

		t_gpu.copy /= (double) 1000;
		t_gpu.exec /= (double) 1000;

		if (warmed_up) {
			const char *style_str = (style == WEDGE_STYLE_ARROW) ? "arrow" : ((style == WEDGE_STYLE_MIXED) ? "mixed" : "outgoing");
			printf("%-60s %16d %16d %16d %16s %16lu %16.6f %16.6f %16.6f %16.6f %16.6f\n",
				graph_filename, graph->numVertices, graph->numEdges, adjacency_matrix_len, style_str, triangles, t_preprocessing, t_gpu.copy, t_gpu.exec, t_gpu.copy + t_gpu.exec, t_cpu);
		} else {
			warmed_up = true;
		}
	}

	free_graph(graph);

	return EXIT_SUCCESS;
}
#include "hip/hip_runtime.h"
/* Vertex-parallel Triangle Counting for big graphs
 * Jeffrey Spaan, Kuan-Hsun Chen, David Bader, Ana-Lucia Varbanescu.
 *
 * Built on the work and code of David Bader. See https://github.com/Bader-Research/triangle-counting/ and https://doi.org/10.1109/HPEC58863.2023.10363539
 *
 * See usage() for instructions.
 * 
 * Assumptions:
 *	- Target GPU is device 0.
 *	- Number of vertices < (uint32_max / 2).
 *	- Number of edges < (uint64_max / 2).
 *	- Number of wedges < uint64_max.
 * 	- Max degree < uint32_max
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <stdbool.h>
#include <sys/time.h>

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_segmented_sort.cuh>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define RESET_DEVICE 0

#define UINT_t uint32_t
#define INT_t int32_t
#define ULONG_t uint64_t

#define max2(a,b) ((a)>(b)?(a):(b))
#define min2(a,b) ((a)<(b)?(a):(b))

static struct timeval	tp;
static struct timezone tzp;

#define get_seconds()	 (gettimeofday(&tp, &tzp), \
												(double)tp.tv_sec + (double)tp.tv_usec / 1000000.0)

#define checkCudaErrors(call)																 						\
	do {																																	\
		hipError_t err = call;																	 						\
		if (err != hipSuccess) {																 						\
			fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,	\
						 hipGetErrorString(err));																	\
			exit(EXIT_FAILURE);																		 						\
		}																												 						\
	} while (0)

enum preprocess_t { PREPROCESS_CPU = 0, PREPROCESS_GPU, PREPROCESS_GPU_CONSTRAINED};

enum wedge_style_t { WEDGE_STYLE_OUTGOING = 0, WEDGE_STYLE_ARROW, WEDGE_STYLE_MIXED};

typedef struct {
	UINT_t numVertices;
	ULONG_t numEdges;
	ULONG_t* rowPtr;
	UINT_t* colInd;
} BIG_GRAPH_TYPE;

typedef struct {
	UINT_t src;
	UINT_t dst;
} edge_t;

typedef struct {
	UINT_t id;
	UINT_t *edges;
	UINT_t num_edges;
} preprocess_vertex_t;

typedef struct {
	double copy;
	double exec;
} GPU_time;

/*********
 *	GPU	*
 *********/

__device__ INT_t linear_search_ULONG_GPU(const UINT_t* list, const ULONG_t start, const ULONG_t end, const UINT_t target) {
	for (ULONG_t i=start; i<end; i++) {
		if (list[i] == target) {
			return i;
		} else if (list[i] > target) {
			break;
		}
	}

	return -1;
}

__device__ INT_t binary_search_ULONG_GPU(const UINT_t* list, const ULONG_t start, const ULONG_t end, const UINT_t target) {
	ULONG_t s=start, e=end, mid;
	while (s < e) {
		mid = (s + e) >> 1;
		if (list[mid] == target)
			return mid;

		if (list[mid] < target)
			s = mid + 1;
		else
			e = mid;
	}
	return -1;
}

__global__ void tc_vertex_outgoing_GPU_kernel(const ULONG_t *g_Ap, const UINT_t *g_Ai, const ULONG_t num_vertices, const ULONG_t vertices_start, const ULONG_t vertices_stop, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const UINT_t v = blockIdx.x * blockDim.x + threadIdx.x + vertices_start;
	ULONG_t count = 0;

	if (v < vertices_stop) {
		ULONG_t vb = g_Ap[v];
		ULONG_t ve = g_Ap[v+1];

		for (ULONG_t i=vb; i<ve; i++) {
			UINT_t w = g_Ai[i];
			ULONG_t wb = g_Ap[w];
			ULONG_t we = g_Ap[w+1];

			for (ULONG_t j=i+1; j<ve; j++) {
				UINT_t u = g_Ai[j];

				if (w >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
					ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-w) * (ULONG_t) ((num_vertices-w)-1)) >> 1)) + u - w - 1;

#if UINT_WIDTH == 32
					bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
#else
					bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
#endif
					if (found) {
						count++;
					}
				} else {
					if (binary_search_ULONG_GPU(g_Ai, wb, we, u) >= 0) {
						count++;
					}
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}

__global__ void tc_vertex_arrow_GPU_kernel(const ULONG_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const ULONG_t vertices_start, const ULONG_t vertices_stop, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const UINT_t v = blockIdx.x * blockDim.x + threadIdx.x + vertices_start;
	ULONG_t count = 0;

	if (v < vertices_stop) {
		ULONG_t vb = g_Ap[v];
		ULONG_t ve = g_Ap[v+1];

		for (ULONG_t i=vb; i<ve; i++) {
			UINT_t w = g_Ai[i];
			ULONG_t wb = g_Ap[w];
			ULONG_t we = g_Ap[w+1];		

			for (ULONG_t j=wb; j<we; j++) {
				UINT_t u = g_Ai[j];

				if (v >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
					ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-v) * (ULONG_t) ((num_vertices-v)-1)) >> 1)) + u - v - 1;

#if UINT_WIDTH == 32
					bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
#else
					bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
#endif
					if (found) {
						count++;
					}
				} else {
					/* Note: in general, searching from vb seems to be faster than from tid+1. */
					if (binary_search_ULONG_GPU(g_Ai, vb, ve, u) >= 0) {
						count++;
					}
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}

__global__ void tc_vertex_mixed_GPU_kernel(const ULONG_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const ULONG_t vertices_start, const ULONG_t vertices_stop, ULONG_t *g_total_count, const UINT_t *g_adjacency_matrix, const UINT_t adjacency_matrix_len, const ULONG_t adjacency_matrix_size) {
	const UINT_t v = blockIdx.x * blockDim.x + threadIdx.x + vertices_start;
	ULONG_t count = 0;

	if (v < vertices_stop) {
		ULONG_t vb = g_Ap[v];
		ULONG_t ve = g_Ap[v+1];

		for (ULONG_t i=vb; i<ve; i++) {
			UINT_t w = g_Ai[i];
			ULONG_t wb = g_Ap[w];
			ULONG_t we = g_Ap[w+1];

			UINT_t size_v = ve-(i+1);
			UINT_t size_w = we-wb;
			
			if (size_v <= size_w) {
				for (ULONG_t j=i+1; j<ve; j++) {
					UINT_t u = g_Ai[j];

					if (w >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
						ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-w) * (ULONG_t) ((num_vertices-w)-1)) >> 1)) + u - w - 1;

#if UINT_WIDTH == 32
						bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
#else
						bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
#endif
						if (found) {
							count++;
						}
					} else {
						if (binary_search_ULONG_GPU(g_Ai, wb, we, u) >= 0) {
							count++;
						}
					}
				}
				
			} else {
				for (ULONG_t j=wb; j<we; j++) {
					UINT_t u = g_Ai[j];

					if (v >= (max2(num_vertices, adjacency_matrix_len) - adjacency_matrix_len)) {
						ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (num_vertices-v) * (ULONG_t) ((num_vertices-v)-1)) >> 1)) + u - v - 1;

#if UINT_WIDTH == 32
						bool found = (g_adjacency_matrix[adjacency_i >> 5] & (1 << (adjacency_i & 31))) > 0;
#else
						bool found = (g_adjacency_matrix[adjacency_i / UINT_WIDTH] & (1 << (adjacency_i % UINT_WIDTH))) > 0;
#endif
						if (found) {
							count++;
						}
					} else {
						if (binary_search_ULONG_GPU(g_Ai, i+1, ve, u) >= 0) {
							count++;
						}
					}
				}
			}
		}
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cg::this_thread_block());

	for (UINT_t i = tile32.size() / 2; i > 0; i /= 2) {
		count += tile32.shfl_down(count, i);
	}

	if (tile32.thread_rank() == 0) atomicAdd((unsigned long long int *) g_total_count, count);
}

/*********
 *	CPU	*
 *********/

static void assert_malloc(const void *ptr) {
	if (ptr==NULL) {
		fprintf(stderr,"ERROR: failed to allocate host memory.\n");
		exit(EXIT_FAILURE);
	}
}

ULONG_t tc_vertex_GPU(const BIG_GRAPH_TYPE *graph, UINT_t adjacency_matrix_len, wedge_style_t style, GPU_time *t) {
	ULONG_t *d_Ap;
	UINT_t *d_Ai;
	UINT_t *d_adjacency_matrix;
	ULONG_t *d_total_count;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));

	/* Store all existing edges of the vertices from n-adjacency_matrix_len to n in an upper triangluar adjacency matrix. */
	ULONG_t adjacency_matrix_size = (((ULONG_t) adjacency_matrix_len) * ((ULONG_t) adjacency_matrix_len - 1)) / 2;
	UINT_t *h_adjacency_matrix = (UINT_t *) calloc(adjacency_matrix_size/UINT_WIDTH, sizeof(UINT_t));
	assert_malloc(h_adjacency_matrix);

	for (UINT_t v=graph->numVertices - min2(graph->numVertices, adjacency_matrix_len); v<graph->numVertices; v++) {
		for (ULONG_t i=graph->rowPtr[v]; i<graph->rowPtr[v+1]; i++) {
			UINT_t w = graph->colInd[i];

			ULONG_t adjacency_i = (adjacency_matrix_size - (((ULONG_t) (graph->numVertices-v) * (ULONG_t) ((graph->numVertices-v)-1)) / 2)) + w - v - 1;

#if UINT_WIDTH == 32
			h_adjacency_matrix[adjacency_i >> 5] |= (1 << (adjacency_i & 31));
#else
			h_adjacency_matrix[adjacency_i / UINT_WIDTH] |= (1 << (adjacency_i % UINT_WIDTH));
#endif
		}
	}

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(ULONG_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_adjacency_matrix, (adjacency_matrix_size/32) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_total_count, 1 * sizeof(ULONG_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(ULONG_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_adjacency_matrix, h_adjacency_matrix, (adjacency_matrix_size/32) * sizeof(UINT_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemset(d_total_count, 0, 1 * sizeof(ULONG_t)));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 64;
	ULONG_t max_blocks = (((ULONG_t) 1 << 31)-1);
	ULONG_t max_vertices = max_blocks * num_threads; // max vertices in a single launch

	for (ULONG_t start=0; start<graph->numVertices; start+=max_vertices) {
		ULONG_t stop = min2(start + max_vertices, graph->numVertices);
		ULONG_t num_blocks = ((stop - start) / (num_threads));

		if (num_blocks < max_blocks)
			num_blocks++;

		dim3 grid(num_blocks, 1, 1);
		dim3 threads(num_threads, 1, 1);

		checkCudaErrors(hipEventRecord(GPU_exec_start));

		if (style == WEDGE_STYLE_OUTGOING)
			tc_vertex_outgoing_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, start, stop, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);
		else if (style == WEDGE_STYLE_ARROW)
			tc_vertex_arrow_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, start, stop, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);
		else if (style == WEDGE_STYLE_MIXED)
			tc_vertex_mixed_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, start, stop, d_total_count, d_adjacency_matrix, adjacency_matrix_len, adjacency_matrix_size);

		checkCudaErrors(hipEventRecord(GPU_exec_stop));
		checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
		checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
		t->exec += GPU_exec_elapsed;
	}

	ULONG_t h_total_count = 0;
	checkCudaErrors(hipMemcpy(&h_total_count, d_total_count, 1 * sizeof(ULONG_t), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_adjacency_matrix));
	checkCudaErrors(hipFree(d_total_count));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_adjacency_matrix);
	
#if RESET_DEVICE
	checkCudaErrors(hipDeviceReset());
#endif
	return h_total_count;
}

void usage() {
	printf("Vertex-parallel Triangle Counting for big graphs\n\n");
	printf("Usage:\n\n");
	printf("Either one of these must be selected:\n");
	printf(" -m <filename>	[Input graph in Matrix Market format]\n");
	printf(" -e <filename>	[Input graph in edge list format]\n");
	printf("Required arguments:\n");
	printf(" -a <num>				[Adjacency matrix length] (must be divisble by 32)\n");
	printf("Optional arguments:\n");
	printf(" -w <num>				[Wedge style, 0:outgoing (default), 1:arrow, 2:mixed]\n");
	printf(" -l <num>				[Loop count]\n");
	printf(" -z							[Input graph is zero-indexed]\n");
	printf(" -p							[Preprocessing style, 0:CPU, 1:GPU, 2:GPU low-memory (default)]\n");
	printf("\n");
	printf("Example:\n");
	printf("./tc_big_graphs_vertex -m ../Amazon0302.mtx -a 8192 -l 10\n");
	exit(EXIT_FAILURE);
}


static int compareInt_t(const void *a, const void *b) {
	UINT_t arg1 = *(const UINT_t *)a;
	UINT_t arg2 = *(const UINT_t *)b;
	if (arg1 < arg2) return -1;
	if (arg1 > arg2) return 1;
	return 0;
}

static int compareEdge_t(const void *a, const void *b) {
	edge_t arg1 = *(const edge_t *) a;
	edge_t arg2 = *(const edge_t *) b;
	if (arg1.src < arg2.src) return -1;
	if (arg1.src > arg2.src) return 1;
	if ((arg1.src == arg2.src) && (arg1.dst < arg2.dst)) return -1;
	if ((arg1.src == arg2.src) && (arg1.dst > arg2.dst)) return 1;
	return 0;
}

static int compare_vertex_degree_ascending(const void *a, const void *b) {
	preprocess_vertex_t arg1 = *(const preprocess_vertex_t *) a;
	preprocess_vertex_t arg2 = *(const preprocess_vertex_t *) b;
	if (arg1.num_edges < arg2.num_edges) return -1;
	if (arg1.num_edges > arg2.num_edges) return 1;
	return 0;
}

struct edge_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&, unsigned int&> operator()(edge_t& key) const {
    return {key.src, key.dst};
  }
};

struct preprocess_vertex_decomposer_t {
  __host__ __device__ ::cuda::std::tuple<unsigned int&> operator()(preprocess_vertex_t& key) const {
    return {key.num_edges};
  }
};

edge_t *sort_edges_GPU(edge_t *d_in, edge_t *d_out, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<edge_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipMalloc((void **) &d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_edges, edge_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

preprocess_vertex_t *sort_vertices_GPU(preprocess_vertex_t *d_in, preprocess_vertex_t *d_out, const UINT_t num_vertices, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<preprocess_vertex_t> d_keys(d_in, d_out);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, num_vertices, preprocess_vertex_decomposer_t{});
		checkCudaErrors(hipFree(d_temp_storage));
		return d_out;
	}
}

UINT_t *sort_colInd_GPU(ULONG_t *d_rowPtr, UINT_t *d_colInd_in, UINT_t *d_colInd_out, const UINT_t num_vertices, const UINT_t num_edges, bool use_double_buffer) {
	std::uint8_t* d_temp_storage{};
	std::size_t temp_storage_bytes{};

	if (use_double_buffer) {
		hipcub::DoubleBuffer<UINT_t> d_keys(d_colInd_in, d_colInd_out);
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_keys.Current();
	} else {
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipMalloc((void **)&d_temp_storage, temp_storage_bytes * sizeof(std::uint8_t)));
		hipcub::DeviceSegmentedSort::SortKeys(d_temp_storage, temp_storage_bytes, d_colInd_in, d_colInd_out, num_edges, num_vertices, d_rowPtr, d_rowPtr + 1);
		checkCudaErrors(hipFree(d_temp_storage));
		return d_colInd_out;
	}
}


BIG_GRAPH_TYPE *read_graph(char *filename, bool matrix_market, bool zero_indexed, preprocess_t preprocess_style) {
	FILE *infile = fopen(filename, "r");
	if (infile == NULL) {
		printf("ERROR: unable to open graph file.\n");
		usage();
	}

	BIG_GRAPH_TYPE *graph = (BIG_GRAPH_TYPE *) malloc(sizeof(BIG_GRAPH_TYPE));
	char line[256];

	/* Skip any header lines */
	do {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	} while (line[0] < '0' || line[0] > '9');

	/* Skip line if the file is in Matrix Market format. We do not use the given vertex/edge counts. */
	if (matrix_market) {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	}

	UINT_t vertex_count = 0;
	ULONG_t edge_count = 0;
	size_t size = 10240;
	edge_t* edges = (edge_t*) malloc(size * sizeof(edge_t));
	assert_malloc(edges);

	UINT_t max_vertex = 0;
	UINT_t v, w;

	if (sscanf(line, "%d %d\n", &v, &w) == 2) {
		do {
			if (edge_count >= size) {
				size += 10240;	
				edge_t *new_edges = (edge_t*) realloc(edges, size * sizeof(edge_t));
				assert_malloc(new_edges);
				edges = new_edges;
			}

			if ((!zero_indexed) && (v == 0 || w == 0)) {
				fprintf(stderr, "ERROR: zero vertex id detected but -z was not set.\n");
				usage();
			}

			v -= (zero_indexed ? 0 : 1);
			w -= (zero_indexed ? 0 : 1);

			/* Remove self-loops. */
			if (v != w) {
				max_vertex = max2(max_vertex, max2(v, w));

				/* v->w */
				edges[edge_count].src = v;
				edges[edge_count].dst = w;
				edge_count++;
				/* w->v */
				edges[edge_count].src = w;
				edges[edge_count].dst = v;
				edge_count++;
			}
		} while (fscanf(infile, "%d %d\n", &v, &w) == 2);
	}

	fclose(infile);

	vertex_count = max_vertex + 1;

	/* Sort edges (in order to remove duplicates). */
	if (preprocess_style != PREPROCESS_CPU) {
		edge_t *d_edges;
		edge_t *d_edges_alt;
		edge_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_edges, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMalloc((void **)&d_edges_alt, edge_count * sizeof(edge_t)));
		checkCudaErrors(hipMemcpy(d_edges, edges, edge_count * sizeof(edge_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, true);
		else
			d_out = sort_edges_GPU(d_edges, d_edges_alt, edge_count, false);

		checkCudaErrors(hipMemcpy(edges, d_out, edge_count * sizeof(edge_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_edges));
		checkCudaErrors(hipFree(d_edges_alt));
	} else {
		qsort(edges, edge_count, sizeof(edge_t), compareEdge_t);
	}	
	
	ULONG_t *rowPtr = (ULONG_t *) calloc(vertex_count+1, sizeof(ULONG_t));
	assert_malloc(rowPtr);

	ULONG_t edge_count_no_dup = 1;

	edge_t lastedge;
	lastedge.src = edges[0].src;
	lastedge.dst = edges[0].dst;

	UINT_t *colInd = (UINT_t *) edges; /* colInd overwrites the edges array. Possible because sizeof(edge_t) > sizeof(UINT_t). */
	colInd[0] = lastedge.dst;
	rowPtr[lastedge.src + 1]++;

	/* Remove duplicate edges. */
	for (ULONG_t i=1; i<edge_count; i++) {
		if (compareEdge_t(&lastedge, &edges[i]) != 0) {
			colInd[edge_count_no_dup++] = edges[i].dst;
			rowPtr[edges[i].src + 1]++;
			lastedge.src = edges[i].src;
			lastedge.dst = edges[i].dst;
		}
	}

	/* Free excess memory from the colInd/edges array. */
	UINT_t *new_colInd = (UINT_t *) realloc(colInd, edge_count_no_dup * sizeof(UINT_t));

	for (UINT_t v=1; v<=vertex_count; v++) {
		rowPtr[v] += rowPtr[v-1];
	}

	graph->numVertices = vertex_count;
	graph->numEdges = edge_count_no_dup;
	graph->rowPtr = rowPtr;
	graph->colInd = new_colInd;

	return graph;
}

BIG_GRAPH_TYPE *preprocess(const BIG_GRAPH_TYPE *original_graph, preprocess_t preprocess_style) {
	preprocess_vertex_t *vertices = (preprocess_vertex_t *) malloc(original_graph->numVertices * sizeof(preprocess_vertex_t));
	assert_malloc(vertices);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		vertices[v].id = v;
		vertices[v].edges = &original_graph->colInd[original_graph->rowPtr[v]];
		vertices[v].num_edges = original_graph->rowPtr[v+1] - original_graph->rowPtr[v];
	}

	if (preprocess_style != PREPROCESS_CPU) {
		preprocess_vertex_t *d_vertices;
		preprocess_vertex_t *d_vertices_alt;
		preprocess_vertex_t *d_out;

		checkCudaErrors(hipMalloc((void **)&d_vertices, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMalloc((void **)&d_vertices_alt, original_graph->numVertices * sizeof(preprocess_vertex_t)));
		checkCudaErrors(hipMemcpy(d_vertices, vertices, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, true);
		else
			d_out = sort_vertices_GPU(d_vertices, d_vertices_alt, original_graph->numVertices, false);

		checkCudaErrors(hipMemcpy(vertices, d_out, original_graph->numVertices * sizeof(preprocess_vertex_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_vertices));
		checkCudaErrors(hipFree(d_vertices_alt));
	} else {
		qsort(vertices, original_graph->numVertices, sizeof(preprocess_vertex_t), compare_vertex_degree_ascending);
	}

	UINT_t *reverse = (UINT_t *) malloc(original_graph->numVertices * sizeof(UINT_t));
	assert_malloc(reverse);

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		reverse[vertices[v].id] = v;
	}

	BIG_GRAPH_TYPE *graph = (BIG_GRAPH_TYPE *) malloc(sizeof(BIG_GRAPH_TYPE));
	assert_malloc(graph);

	graph->numVertices = original_graph->numVertices;
	graph->numEdges = original_graph->numEdges/2;

	graph->rowPtr = (ULONG_t*) malloc((graph->numVertices + 1) * sizeof(ULONG_t));
	assert_malloc(graph->rowPtr);
	graph->colInd = (UINT_t*) malloc(graph->numEdges * sizeof(UINT_t));
	assert_malloc(graph->colInd);

	ULONG_t edge_count = 0;

	graph->rowPtr[0] = 0;

	for (UINT_t v=0; v<original_graph->numVertices; v++) {
		UINT_t new_degree = 0;

		for (UINT_t j=0; j<vertices[v].num_edges; j++) {
			UINT_t w = vertices[v].edges[j];
			UINT_t w_new = reverse[w];

			if (w_new > v) {
				graph->colInd[edge_count++] = w_new;
				new_degree++;
			}
		}

		graph->rowPtr[v+1] = graph->rowPtr[v] + new_degree;

		if (preprocess_style == PREPROCESS_CPU) {
			qsort(&graph->colInd[graph->rowPtr[v]], new_degree, sizeof(UINT_t), compareInt_t);
		}
	}

	free(vertices);
	free(reverse);

	if (preprocess_style != PREPROCESS_CPU) {
		ULONG_t *d_rowPtr;
		UINT_t *d_colInd;
		UINT_t *d_colInd_alt;
		UINT_t *d_colInd_out;

		checkCudaErrors(hipMalloc((void **)&d_rowPtr, (graph->numVertices+1) * sizeof(ULONG_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMalloc((void **)&d_colInd_alt, graph->numEdges * sizeof(UINT_t)));
		checkCudaErrors(hipMemcpy(d_rowPtr, graph->rowPtr, (graph->numVertices+1) * sizeof(ULONG_t), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_colInd, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));

		if (preprocess_style == PREPROCESS_GPU_CONSTRAINED)
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, true);
		else
			d_colInd_out = sort_colInd_GPU(d_rowPtr, d_colInd, d_colInd_alt, graph->numVertices, graph->numEdges, false);

		checkCudaErrors(hipMemcpy(graph->colInd, d_colInd_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_rowPtr));
		checkCudaErrors(hipFree(d_colInd));
		checkCudaErrors(hipFree(d_colInd_alt));
	}

	return graph;
}

void free_graph(BIG_GRAPH_TYPE *graph) {
	free(graph->rowPtr);
	free(graph->colInd);
	free(graph);
}

int main(int argc, char **argv) {
	char *graph_filename = NULL;
	bool graph_mm = false;
	bool graph_zero_indexed = false;
	UINT_t loop_cnt = 1;

	/* Default: use lightweight GPU-based preprocessing (worst case ~ m*8 device memory). */
	preprocess_t preprocess_style = PREPROCESS_GPU_CONSTRAINED;

	wedge_style_t style = WEDGE_STYLE_OUTGOING;
	UINT_t adjacency_matrix_len = 0;

	while ((argc > 1) && (argv[1][0] == '-')) {
		switch (argv[1][1]) {
			case 'm':
				graph_mm = true;
			case 'e':
				if (argc < 3) usage();
				graph_filename = argv[2];
				if (graph_filename == NULL) usage();
				argv+=2;
				argc-=2;
				break;
			case 'z':
				graph_zero_indexed = true;
				argv++;
				argc--;
				break;
			case 'w':
				if (argc < 3) usage();
				if (atoi(argv[2]) < WEDGE_STYLE_OUTGOING || atoi(argv[2]) > WEDGE_STYLE_MIXED) usage();
				style = (wedge_style_t) atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
			case 'a':
				/* Adjacency matrix lengths:
				 *	 8192 ~= 4MiB
				 *	 16384 ~= 16MiB
				 *	 32768 ~= 64MiB
				 *	 65536 ~= 265MiB
				 *	 131072 ~= 1024MiB
				 */
				if (argc < 3) usage();
				adjacency_matrix_len = atoi(argv[2]);
				if (adjacency_matrix_len % 32 != 0) usage();
				argv+=2;
				argc-=2;
				break;
			case 'l':
				if (argc < 3) usage();
				loop_cnt = atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
			case 'p':
				if (argc < 3) usage();
				if (atoi(argv[2]) < PREPROCESS_CPU || atoi(argv[2]) > PREPROCESS_GPU_CONSTRAINED) usage();
				preprocess_style = (preprocess_t) atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
		}
	}

	if (graph_filename == NULL) usage();

	BIG_GRAPH_TYPE *original_graph = read_graph(graph_filename, graph_mm, graph_zero_indexed, preprocess_style);
	double t_preprocessing = get_seconds();
	BIG_GRAPH_TYPE *graph = preprocess(original_graph, preprocess_style);
	t_preprocessing = get_seconds() - t_preprocessing;
	free_graph(original_graph);

	printf("%-60s %16s %22s %16s %16s %22s %16s %16s %16s %16s %16s\n",
		"graph", "n", "m", "a", "wedge style", "triangles", "prepro (s)", "GPU copy (s)", "GPU exec (s)", "GPU total (s)", "CPU+GPU (s)");

	bool warmed_up = false;

	for (UINT_t i=0; i<(loop_cnt+1); i++) {
		double t_cpu = get_seconds();
		GPU_time t_gpu = { .copy=0.0, .exec=0.0 };

		ULONG_t triangles = tc_vertex_GPU(graph, adjacency_matrix_len, style, &t_gpu);

		t_cpu = get_seconds() - t_cpu;

		t_gpu.copy /= (double) 1000;
		t_gpu.exec /= (double) 1000;

		if (warmed_up) {
			const char *style_str = (style == WEDGE_STYLE_ARROW) ? "arrow" : ((style == WEDGE_STYLE_MIXED) ? "mixed" : "outgoing");
			printf("%-60s %16d %22lu %16d %16s %22lu %16.6f %16.6f %16.6f %16.6f %16.6f\n",
				graph_filename, graph->numVertices, graph->numEdges, adjacency_matrix_len, style_str, triangles, t_preprocessing, t_gpu.copy, t_gpu.exec, t_gpu.copy + t_gpu.exec, t_cpu);
		} else {
			warmed_up = true;
		}
	}

	free_graph(graph);

	return EXIT_SUCCESS;
}
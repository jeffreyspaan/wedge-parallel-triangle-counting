/* Edge retrieval strategy experiment
 * Jeffrey Spaan, Ana-Lucia Varbanescu, Kuan Chen.
 *
 * Built on the work and code of David Bader. See https://github.com/Bader-Research/triangle-counting/ and https://doi.org/10.1109/HPEC58863.2023.10363539
 *
 * See usage() for instructions.
 * 
 * Assumptions:
 *	- Target GPU is device 0.
 *	- Number of vertices < (uint32_max / 2).
 *	- Number of edges < (uint32__max / 2).
 *	- Number of wedges < (2^31 - 1) * 128 * spread.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <stdbool.h>
#include <sys/time.h>

#define CHECK_BOUNDS 1
#define RESET_DEVICE 0

#define BINSEARCH_CONSTANT_LEVELS 12
#define BINSEARCH_CONSTANT_CACHE_SIZE ((1 << BINSEARCH_CONSTANT_LEVELS) - 1) // 2^levels - 1

#define UINT_t uint32_t
#define INT_t int32_t
#define ULONG_t uint64_t

#define max2(a,b) ((a)>(b)?(a):(b))
#define min2(a,b) ((a)<(b)?(a):(b))

static struct timeval	tp;
static struct timezone tzp;

#define get_seconds()	 (gettimeofday(&tp, &tzp), \
												(double)tp.tv_sec + (double)tp.tv_usec / 1000000.0)

#define checkCudaErrors(call)																						\
	do {																																	\
		hipError_t err = call;																	 						\
		if (err != hipSuccess) {																 						\
			fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,	\
						 hipGetErrorString(err));																	\
			exit(EXIT_FAILURE);																		 						\
		}																												 						\
	} while (0)

typedef struct {
	UINT_t numVertices;
	UINT_t numEdges;
	UINT_t* rowPtr;
	UINT_t* colInd;
} GRAPH_TYPE;

typedef struct {
	UINT_t src;
	UINT_t dst;
} edge_t;

typedef struct {
	UINT_t id;
	UINT_t new_id;
	UINT_t num_edges;
	UINT_t *edges;
} preprocess_vertex_t;

typedef struct {
	double copy;
	double exec;
} GPU_time;

/*********
 *	GPU	*
 *********/

__constant__ UINT_t c_binary_search_cache[BINSEARCH_CONSTANT_CACHE_SIZE];

__device__ INT_t binary_search_GPU(const UINT_t* list, const UINT_t start, const UINT_t end, const UINT_t target) {
	UINT_t s=start, e=end, mid;
	while (s < e) {
		mid = (s + e) >> 1;
		if (list[mid] == target)
			return mid;

		if (list[mid] < target)
			s = mid + 1;
		else
			e = mid;
	}
	return -1;
}


__device__ UINT_t binary_search_closest_GPU(const UINT_t* list, const UINT_t start, const UINT_t end, const UINT_t target) {
	/* Finds the index of the rightmost closest value smaller or equal than target, e.g.,
	 * for target 1 and list=[0,0,0,2,2,2] it returns 2,
	 * for target 2 and list=[0,0,0,2,2,2] it returns 5.
	 * Assumes list[0]=0
	 * Assumes end-1 <= UINT_MAX/2
	 */

	UINT_t s=start, e=end, mid;
	while (s < e) {
		mid = (s + e) >> 1;

		if (list[mid] < target+1) {
			s = mid + 1;
		} else {
			e = mid;
		}
	}
	
	return max2(start, (s > 0) ? s-1: 0);
}

__device__ UINT_t binary_search_closest_constant_GPU(const UINT_t *list, const UINT_t start, const UINT_t end, const UINT_t target) {
	/* Finds the index of the rightmost closest value smaller or equal than target.
	 * Uses constant memory for the first BINSEARCH_CONSTANT_LEVELS levels.
	 */
	UINT_t mid;

	UINT_t g_s = start;
	UINT_t g_e = end;
	UINT_t g_mid;

	UINT_t c_index = 0;

	#pragma unroll
	for (UINT_t iter=0; iter<BINSEARCH_CONSTANT_LEVELS; iter++) {
		mid = c_binary_search_cache[c_index];
		g_mid = (g_s+g_e) >> 1;

		c_index *= 2;
		c_index += 1;

		if (mid < target+1) {
			c_index += 1;
			g_s = g_mid+1;
		} else {
			g_e = g_mid;
		}
	}

	g_s = max2(start, (g_s > 0) ? g_s-1 : 0);
	return binary_search_closest_GPU(list, g_s, g_e, target);
}

typedef struct {
	UINT_t src;
} edge_src_t;

__global__ void edge_get_edgelist_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const edge_t *g_edges, const UINT_t num_vertices, const UINT_t num_edges, UINT_t *g_out) {
	const UINT_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid < num_edges) {
		UINT_t v = g_edges[tid].src;
		UINT_t w = g_edges[tid].dst;

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		if (binary_search_GPU(g_Ai, wb, we, v) >= 0) {
			g_out[tid] = 1;
		} else {
			g_out[tid] = 0;
		}
	}
}

__global__ void edge_get_edgelist_src_only_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const edge_src_t *g_edges_src, const UINT_t num_vertices, const UINT_t num_edges, UINT_t *g_out) {
	const UINT_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid < num_edges) {
		UINT_t v = g_edges_src[tid].src;
		UINT_t w = g_Ai[tid];

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		if (binary_search_GPU(g_Ai, wb, we, v) >= 0) {
			g_out[tid] = 1;
		} else {
			g_out[tid] = 0;
		}
	}
}

__global__ void edge_get_binary_search_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const UINT_t num_edges, UINT_t *g_out) {
	const UINT_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid < num_edges) {
		UINT_t w = g_Ai[tid];
		UINT_t v = binary_search_closest_GPU(g_Ap, 0, num_vertices, tid);

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		if (binary_search_GPU(g_Ai, wb, we, v) >= 0) {
			g_out[tid] = 1;
		} else {
			g_out[tid] = 0;
		}
	}
}

__global__ void edge_get_binary_search_cached_GPU_kernel(const UINT_t *g_Ap, const UINT_t *g_Ai, const UINT_t num_vertices, const UINT_t num_edges, UINT_t *g_out) {
	const UINT_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid < num_edges) {
		UINT_t w = g_Ai[tid];
		UINT_t v = binary_search_closest_constant_GPU(g_Ap, 0, num_vertices, tid);

		UINT_t wb = g_Ap[w];
		UINT_t we = g_Ap[w+1];

		if (binary_search_GPU(g_Ai, wb, we, v) >= 0) {
			g_out[tid] = 1;
		} else {
			g_out[tid] = 0;
		}
	}
}

/*********
 *	CPU	*
 *********/

static void assert_malloc(const void *ptr) {
	if (ptr==NULL) {
		fprintf(stderr,"ERROR: failed to allocate host memory.\n");
		exit(EXIT_FAILURE);
	}
}

void build_binary_search_cache(UINT_t *src, UINT_t *cache, UINT_t level, UINT_t max_level, UINT_t i, UINT_t s, UINT_t e) {
	if (level < max_level) {
		UINT_t mid = (s + e) / 2;
		cache[i] = src[mid];
		build_binary_search_cache(src, cache, level+1, max_level, i*2 + 1, s, mid);
		build_binary_search_cache(src, cache, level+1, max_level, i*2 + 2, mid+1, e);
	}
}

void edge_get_edgelist_GPU(const GRAPH_TYPE *graph, GPU_time *t) {
	UINT_t *d_Ap;
	UINT_t *d_Ai;
	edge_t *d_edges;
	UINT_t *d_out;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));

	edge_t *h_edges = (edge_t *) malloc(graph->numEdges * sizeof(edge_t));
	UINT_t edge_ctr = 0;

	for (UINT_t v=0; v<graph->numVertices; v++) {
		for (UINT_t i=graph->rowPtr[v]; i<graph->rowPtr[v+1]; i++) {
			UINT_t w = graph->colInd[i];
			h_edges[edge_ctr].src = v;
			h_edges[edge_ctr].dst = w;
			edge_ctr++;
		}
	}

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_edges, graph->numEdges * sizeof(edge_t)));
	checkCudaErrors(hipMalloc((void **)&d_out, graph->numEdges * sizeof(UINT_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_edges, h_edges, graph->numEdges * sizeof(edge_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 128;
	ULONG_t num_blocks = (graph->numEdges / num_threads) + 1;

	if (num_blocks > (((ULONG_t) 1 << 31)-1)) {
		fprintf(stderr, "ERROR: maximum grid size reached.\n");
		exit(EXIT_FAILURE);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipEventRecord(GPU_exec_start));

	edge_get_edgelist_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, d_edges, graph->numVertices, graph->numEdges, d_out);

	checkCudaErrors(hipEventRecord(GPU_exec_stop));
	checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
	t->exec += GPU_exec_elapsed;

	UINT_t *h_out = (UINT_t *) malloc(graph->numEdges * sizeof(UINT_t));
	checkCudaErrors(hipMemcpy(h_out, d_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));

	/* Confirm (w,v) does not exist (=0) for every edge (v,w). */
	for (UINT_t i=0; i<graph->numEdges; i++) {
		if (h_out[i] != 0) {
			fprintf(stderr, "Nonzero found.\n");
			exit(EXIT_FAILURE);
		}
	}

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_edges));
	checkCudaErrors(hipFree(d_out));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_edges);
	free(h_out);
	
#if RESET_DEVICE
	checkCudaErrors(cudaDeviceReset());
#endif
}

void edge_get_edgelist_src_only_GPU(const GRAPH_TYPE *graph, GPU_time *t) {
	UINT_t *d_Ap;
	UINT_t *d_Ai;
	edge_src_t *d_edges_src;
	UINT_t *d_out;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));

	edge_src_t *h_edges_src = (edge_src_t *) malloc(graph->numEdges * sizeof(edge_src_t));
	UINT_t edge_ctr = 0;

	for (UINT_t v=0; v<graph->numVertices; v++) {
		for (UINT_t i=graph->rowPtr[v]; i<graph->rowPtr[v+1]; i++) {
			h_edges_src[edge_ctr].src = v;
			edge_ctr++;
		}
	}

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_edges_src, graph->numEdges * sizeof(edge_src_t)));
	checkCudaErrors(hipMalloc((void **)&d_out, graph->numEdges * sizeof(UINT_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_edges_src, h_edges_src, graph->numEdges * sizeof(edge_src_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 128;
	ULONG_t num_blocks = (graph->numEdges / num_threads) + 1;

	if (num_blocks > (((ULONG_t) 1 << 31)-1)) {
		fprintf(stderr, "ERROR: maximum grid size reached.\n");
		exit(EXIT_FAILURE);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipEventRecord(GPU_exec_start));

	edge_get_edgelist_src_only_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, d_edges_src, graph->numVertices, graph->numEdges, d_out);

	checkCudaErrors(hipEventRecord(GPU_exec_stop));
	checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
	t->exec += GPU_exec_elapsed;

	UINT_t *h_out = (UINT_t *) malloc(graph->numEdges * sizeof(UINT_t));
	checkCudaErrors(hipMemcpy(h_out, d_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));

	/* Confirm (w,v) does not exist (=0) for every edge (v,w). */
	for (UINT_t i=0; i<graph->numEdges; i++) {
		if (h_out[i] != 0) {
			fprintf(stderr, "Nonzero found.\n");
			exit(EXIT_FAILURE);
		}
	}

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_edges_src));
	checkCudaErrors(hipFree(d_out));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_edges_src);
	free(h_out);
	
#if RESET_DEVICE
	checkCudaErrors(cudaDeviceReset());
#endif
}

void edge_get_binary_search_GPU(const GRAPH_TYPE *graph, GPU_time *t) {
	UINT_t *d_Ap;
	UINT_t *d_Ai;
	UINT_t *d_out;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_out, graph->numEdges * sizeof(UINT_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 128;
	ULONG_t num_blocks = (graph->numEdges / num_threads) + 1;

	if (num_blocks > (((ULONG_t) 1 << 31)-1)) {
		fprintf(stderr, "ERROR: maximum grid size reached.\n");
		exit(EXIT_FAILURE);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipEventRecord(GPU_exec_start));

	edge_get_binary_search_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, graph->numEdges, d_out);

	checkCudaErrors(hipEventRecord(GPU_exec_stop));
	checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
	t->exec += GPU_exec_elapsed;

	UINT_t *h_out = (UINT_t *) malloc(graph->numEdges * sizeof(UINT_t));
	checkCudaErrors(hipMemcpy(h_out, d_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));

	/* Confirm (w,v) does not exist (=0) for every edge (v,w). */
	for (UINT_t i=0; i<graph->numEdges; i++) {
		// if (h_out[i] != 0) {
		// 	fprintf(stderr, "Nonzero found.\n");
		// 	exit(EXIT_FAILURE);
		// }
	}

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_out));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_out);

#if RESET_DEVICE
	checkCudaErrors(cudaDeviceReset());
#endif
}

void edge_get_binary_search_cached_GPU(const GRAPH_TYPE *graph, GPU_time *t) {
	UINT_t *d_Ap;
	UINT_t *d_Ai;
	UINT_t *d_out;

	hipEvent_t GPU_copy_start, GPU_copy_stop, GPU_exec_start, GPU_exec_stop;
	float GPU_copy_elapsed, GPU_exec_elapsed;
	checkCudaErrors(hipEventCreate(&GPU_copy_start));
	checkCudaErrors(hipEventCreate(&GPU_copy_stop));
	checkCudaErrors(hipEventCreate(&GPU_exec_start));
	checkCudaErrors(hipEventCreate(&GPU_exec_stop));
	
	UINT_t *h_rowPtr_cache = (UINT_t *) malloc(BINSEARCH_CONSTANT_CACHE_SIZE * sizeof(UINT_t));
	assert_malloc(h_rowPtr_cache);
	build_binary_search_cache(graph->rowPtr, h_rowPtr_cache, 0, BINSEARCH_CONSTANT_LEVELS, 0, 0, graph->numVertices);

	checkCudaErrors(hipEventRecord(GPU_copy_start));

	checkCudaErrors(hipMalloc((void **)&d_Ap, (graph->numVertices + 1) * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_Ai, graph->numEdges * sizeof(UINT_t)));
	checkCudaErrors(hipMalloc((void **)&d_out, graph->numEdges * sizeof(UINT_t)));

	checkCudaErrors(hipMemcpy(d_Ap, graph->rowPtr, (graph->numVertices + 1) * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Ai, graph->colInd, graph->numEdges * sizeof(UINT_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_binary_search_cache), h_rowPtr_cache, BINSEARCH_CONSTANT_CACHE_SIZE * sizeof(UINT_t)));

	checkCudaErrors(hipEventRecord(GPU_copy_stop));
	checkCudaErrors(hipEventSynchronize(GPU_copy_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_copy_elapsed, GPU_copy_start, GPU_copy_stop));
	t->copy += GPU_copy_elapsed;

	UINT_t num_threads = 128;
	ULONG_t num_blocks = (graph->numEdges / num_threads) + 1;

	if (num_blocks > (((ULONG_t) 1 << 31)-1)) {
		fprintf(stderr, "ERROR: maximum grid size reached.\n");
		exit(EXIT_FAILURE);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	checkCudaErrors(hipEventRecord(GPU_exec_start));

	edge_get_binary_search_cached_GPU_kernel<<<grid, threads>>>(d_Ap, d_Ai, graph->numVertices, graph->numEdges, d_out);

	checkCudaErrors(hipEventRecord(GPU_exec_stop));
	checkCudaErrors(hipEventSynchronize(GPU_exec_stop));
	checkCudaErrors(hipEventElapsedTime(&GPU_exec_elapsed, GPU_exec_start, GPU_exec_stop));
	t->exec += GPU_exec_elapsed;

	UINT_t *h_out = (UINT_t *) malloc(graph->numEdges * sizeof(UINT_t));
	checkCudaErrors(hipMemcpy(h_out, d_out, graph->numEdges * sizeof(UINT_t), hipMemcpyDeviceToHost));

	/* Confirm (w,v) does not exist (=0) for every edge (v,w). */
	for (UINT_t i=0; i<graph->numEdges; i++) {
		// if (h_out[i] != 0) {
		// 	fprintf(stderr, "Nonzero found.\n");
		// 	exit(EXIT_FAILURE);
		// }
	}

	checkCudaErrors(hipFree(d_Ap));
	checkCudaErrors(hipFree(d_Ai));
	checkCudaErrors(hipFree(d_out));

	checkCudaErrors(hipEventDestroy(GPU_copy_start));
	checkCudaErrors(hipEventDestroy(GPU_copy_stop));
	checkCudaErrors(hipEventDestroy(GPU_exec_start));
	checkCudaErrors(hipEventDestroy(GPU_exec_stop));

	free(h_rowPtr_cache);
	free(h_out);

#if RESET_DEVICE
	checkCudaErrors(cudaDeviceReset());
#endif
}

void usage() {
	printf("Edge retrieval strategy experiment\n\n");
	printf("Usage:\n\n");
	printf("Either one of these must be selected:\n");
	printf(" -m <filename>		[Input graph in Matrix Market format]\n");
	printf(" -e <filename>		[Input graph in edge list format]\n");
	printf("Optional arguments:\n");
	printf(" -l <num>					[Loop count]\n");
	printf(" -z								[Input graph is zero-indexed]\n");
	printf("\n");
	printf("Example:\n");
	printf("./edge_get -m ../Amazon0302.mtx -l 10\n");
	exit(EXIT_FAILURE);
}

static int compareInt_t(const void *a, const void *b) {
	UINT_t arg1 = *(const UINT_t *)a;
	UINT_t arg2 = *(const UINT_t *)b;
	if (arg1 < arg2) return -1;
	if (arg1 > arg2) return 1;
	return 0;
}

static int compareEdge_t(const void *a, const void *b) {
	edge_t arg1 = *(const edge_t *) a;
	edge_t arg2 = *(const edge_t *) b;
	if (arg1.src < arg2.src) return -1;
	if (arg1.src > arg2.src) return 1;
	if ((arg1.src == arg2.src) && (arg1.dst < arg2.dst)) return -1;
	if ((arg1.src == arg2.src) && (arg1.dst > arg2.dst)) return 1;
	return 0;
}

static int compare_vertex_degree_ascending(const void *a, const void *b) {
	preprocess_vertex_t arg1 = *(const preprocess_vertex_t *) a;
	preprocess_vertex_t arg2 = *(const preprocess_vertex_t *) b;
	if (arg1.num_edges < arg2.num_edges) return -1;
	if (arg1.num_edges > arg2.num_edges) return 1;
	return 0;
}

GRAPH_TYPE *read_graph(char *filename, bool matrix_market, bool zero_indexed) {
	FILE *infile = fopen(filename, "r");
	if (infile == NULL) {
		printf("ERROR: unable to open graph file.\n");
		usage();
	}

	GRAPH_TYPE *graph = (GRAPH_TYPE *) malloc(sizeof(GRAPH_TYPE));
	char line[256];

	/* Skip any header lines */
	do {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	} while (line[0] < '0' || line[0] > '9');

	/* Skip line if the file is in Matrix Market format. We do not use the given vertex/edge counts. */
	if (matrix_market) {
		if (fgets(line, sizeof(line), infile) == NULL) usage();
	}

	UINT_t vertex_count = 0;
	UINT_t edge_count = 0;
	size_t size = 10240;
	edge_t* edges = (edge_t*) malloc(size * sizeof(edge_t));
	assert_malloc(edges);

	UINT_t max_vertex = 0;
	UINT_t v, w;

	if (sscanf(line, "%d %d\n", &v, &w) == 2) {
		do {
			if (edge_count >= size) {
				size += 10240;
				edge_t *new_edges = (edge_t*) realloc(edges, size * sizeof(edge_t));
				assert_malloc(new_edges);
				edges = new_edges;
			}

			if ((!zero_indexed) && (v == 0 || w == 0)) {
				fprintf(stderr, "ERROR: zero vertex id detected but -z was not set.\n");
				usage();
			}

			v -= (zero_indexed ? 0 : 1);
			w -= (zero_indexed ? 0 : 1);

			/* Remove self-loops. */
			if (v != w) {
				max_vertex = max2(max_vertex, max2(v, w));

				/* v->w */
				edges[edge_count].src = v;
				edges[edge_count].dst = w;
				edge_count++;
				/* w->v */
				edges[edge_count].src = w;
				edges[edge_count].dst = v;
				edge_count++;
			}
		} while (fscanf(infile, "%d %d\n", &v, &w) == 2);
	}

	fclose(infile);

	vertex_count = max_vertex + 1;

	/* Sort edges (in order to remove duplicates). */
	qsort(edges, edge_count, sizeof(edge_t), compareEdge_t);

	UINT_t *rowPtr = (UINT_t *) calloc(vertex_count+1, sizeof(UINT_t));
	assert_malloc(rowPtr);

	UINT_t edge_count_no_dup = 1;

	edge_t lastedge;
	lastedge.src = edges[0].src;
	lastedge.dst = edges[0].dst;

	UINT_t *colInd = (UINT_t *) edges; /* colInd overwrites the edges array. Possible because sizeof(edge_t) > sizeof(UINT_t). */
	colInd[0] = lastedge.dst;
	rowPtr[lastedge.src + 1]++;

	/* Remove duplicate edges. */
	for (UINT_t i=1; i<edge_count; i++) {
		if (compareEdge_t(&lastedge, &edges[i]) != 0) {
			colInd[edge_count_no_dup++] = edges[i].dst;
			rowPtr[edges[i].src + 1]++;
			lastedge.src = edges[i].src;
			lastedge.dst = edges[i].dst;
		}
	}

	/* Free excess memory from the colInd/edges array. */
	UINT_t *new_colInd = (UINT_t *) realloc(colInd, edge_count_no_dup * sizeof(UINT_t));

	for (UINT_t v=1; v<=vertex_count; v++) {
		rowPtr[v] += rowPtr[v-1];
	}

	graph->numVertices = vertex_count;
	graph->numEdges = edge_count_no_dup;
	graph->rowPtr = rowPtr;
	graph->colInd = new_colInd;

	return graph;
}


GRAPH_TYPE *preprocess(const GRAPH_TYPE *original_graph) {
	UINT_t n = original_graph->numVertices;
	UINT_t new_n = 0;
	UINT_t max_degree = 0;

	UINT_t *a = (UINT_t *) calloc(n, sizeof(UINT_t));
	assert_malloc(a);

	for (UINT_t v=0; v<n; v++) {
		max_degree = max2(max_degree, original_graph->rowPtr[v+1] - original_graph->rowPtr[v]);

		for (UINT_t j=original_graph->rowPtr[v]; j<original_graph->rowPtr[v+1]; j++) {
			UINT_t u = original_graph->colInd[j];
			a[v] = 1;
			a[u] = 1;
		}
	}

	for (UINT_t i=0; i<n; i++) {
		if (a[i] == 1) {
			a[i] = new_n++;
		}
	}

	preprocess_vertex_t *vertices = (preprocess_vertex_t *) malloc(new_n * sizeof(preprocess_vertex_t));
	assert_malloc(vertices);

	for (UINT_t v=0; v<new_n; v++) {
		vertices[v].id = v;
		vertices[v].edges = NULL;
		vertices[v].num_edges = 0;
	}

	for (UINT_t v=0; v<n; v++) {
		for (UINT_t j=original_graph->rowPtr[v]; j<original_graph->rowPtr[v+1]; j++) {
			UINT_t u = original_graph->colInd[j];
			vertices[a[v]].edges = (UINT_t *) realloc(vertices[a[v]].edges, (vertices[a[v]].num_edges + 1) * sizeof(UINT_t));
			vertices[a[v]].edges[vertices[a[v]].num_edges++] = a[u];
		}
	}

	free(a);

	UINT_t *reverse = (UINT_t *) malloc(new_n * sizeof(UINT_t));

	qsort(vertices, new_n, sizeof(preprocess_vertex_t), compare_vertex_degree_ascending);

	for (UINT_t v=0; v<new_n; v++) {
		reverse[vertices[v].id] = v;
	}

	for (UINT_t v=0; v<new_n; v++) {
		vertices[v].id = v;
		UINT_t new_num_edges = 0;

		for (INT_t j=0; j<vertices[v].num_edges; j++) {
			UINT_t w = vertices[v].edges[j];
			UINT_t w_new = reverse[w];

			if (w_new > v) {
				vertices[v].edges[new_num_edges++] = w_new;
			}
		}

		vertices[v].num_edges = new_num_edges;
	}

	free(reverse);

	GRAPH_TYPE *graph = (GRAPH_TYPE *) malloc(sizeof(GRAPH_TYPE));
	assert_malloc(graph);

	graph->numVertices = new_n;
	graph->numEdges = original_graph->numEdges/2;

	graph->rowPtr = (UINT_t*) malloc((graph->numVertices + 1) * sizeof(UINT_t));
	assert_malloc(graph->rowPtr);
	graph->colInd = (UINT_t*) malloc(graph->numEdges * sizeof(UINT_t));
	assert_malloc(graph->colInd);

	graph->rowPtr[0] = 0;

	for (UINT_t v=0; v<new_n; v++) {
		graph->rowPtr[v+1] = graph->rowPtr[v] + vertices[v].num_edges;

		for (UINT_t j=0; j<vertices[v].num_edges; j++) {
			graph->colInd[graph->rowPtr[v] + j] = vertices[v].edges[j];
		}

		qsort(&graph->colInd[graph->rowPtr[v]], vertices[v].num_edges, sizeof(UINT_t), compareInt_t);
	}

	for (UINT_t v=0; v<new_n; v++)
		free(vertices[v].edges);
	free(vertices);

	return graph;
}

void free_graph(GRAPH_TYPE *graph) {
	free(graph->rowPtr);
	free(graph->colInd);
	free(graph);
}

int main(int argc, char **argv) {
	char *graph_filename = NULL;
	bool graph_mm = false;
	bool graph_zero_indexed = false;
	UINT_t loop_cnt = 1;

	while ((argc > 1) && (argv[1][0] == '-')) {
		switch (argv[1][1]) {
			case 'm':
				graph_mm = true;
			case 'e':
				if (argc < 3) usage();
				graph_filename = argv[2];
				if (graph_filename == NULL) usage();
				argv+=2;
				argc-=2;
				break;
			case 'z':
				graph_zero_indexed = true;
				argv++;
				argc--;
				break;
			case 'l':
				if (argc < 3) usage();
				loop_cnt = atoi(argv[2]);
				argv+=2;
				argc-=2;
				break;
		}
	}

	if (graph_filename == NULL) usage();

	GRAPH_TYPE *original_graph = read_graph(graph_filename, graph_mm, graph_zero_indexed);
	GRAPH_TYPE *graph = preprocess(original_graph);
	free_graph(original_graph);

	printf("%-60s %16s %16s %24s %16s %16s %16s %16s\n",
		"graph", "n", "m", "retrieval", "GPU copy (s)", "GPU exec (s)", "GPU total (s)", "CPU+GPU (s)");

	const char *strats_names[4] = {"edge_list", "edge_list_src", "binary_search", "binary_search_cached"};
	void (*(strats_functions[4]))(const GRAPH_TYPE *, GPU_time *) = {edge_get_edgelist_GPU, edge_get_edgelist_src_only_GPU, edge_get_binary_search_GPU, edge_get_binary_search_cached_GPU};

	for (UINT_t strat=0; strat<4; strat++) {
		bool warmed_up = false;

		for (UINT_t i=0; i<(loop_cnt+1); i++) {
			double t_cpu = get_seconds();
			GPU_time t_gpu = { .copy=0.0, .exec=0.0 };

			strats_functions[strat](graph, &t_gpu);

			t_cpu = get_seconds() - t_cpu;

			t_gpu.copy /= (double) 1000;
			t_gpu.exec /= (double) 1000;

			if (warmed_up) {
						printf("%-60s %16d %16d %24s %16.6f %16.6f %16.6f %16.6f\n",
					graph_filename, graph->numVertices, graph->numEdges, strats_names[strat], t_gpu.copy, t_gpu.exec, t_gpu.copy + t_gpu.exec, t_cpu);
			} else {
				warmed_up = true;
			}
		}
	}

	free_graph(graph);

	return EXIT_SUCCESS;
}